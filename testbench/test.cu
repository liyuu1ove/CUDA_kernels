﻿#include <hip/hip_runtime.h>
#include "../hello/hello_GPU.h"
int main(){
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start); // 记录起始时间
hello_GPU(); // 执行内核
hipEventRecord(stop);  // 记录结束时间
hipEventSynchronize(stop);
float time_ms;
hipEventElapsedTime(&time_ms, start, stop);
printf("Kernel time: %f ms\n", time_ms);

hipEventDestroy(start);
hipEventDestroy(stop);
}