#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel.h"

__global__ void my_kernel() {
    printf("Hello from GPU thread %d\n", threadIdx.x);
}

void hello_warpper() {
    // 启动GPU核函数
    my_kernel<<<1, 5>>>();  // 1个block，5个线程
    hipDeviceSynchronize(); // 等待GPU完成
}