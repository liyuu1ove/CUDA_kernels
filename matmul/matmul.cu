#include "hip/hip_runtime.h"
#include "matmul.cuh"

__global__ void matmul_v1(float* A,float* B,float* C,int M,int N,int K){
    int x=blockIdx.x*blockDim.x+threadIdx.x;
    int y=blockIdx.y*blockDim.y+threadIdx.y;
    if(row<M&&col<K){
        float temp=0;
    for(int i=0;i<N;i++){
        temp+=A[row*N+i]*B[i*K+col];
        }
        C[row*K+col]=temp;
    }
}
template<int BLOCKSIZE>
__global__ void matmul_v2(float* A,float* B,float* C,int M,int N,int K){

}
