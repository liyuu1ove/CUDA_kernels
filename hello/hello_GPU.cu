#include "hip/hip_runtime.h"
﻿#include "hello_GPU.h"



__global__ void hello_GPU_1D(){
    int index=threadIdx.x+blockIdx.x*blockDim.x;
    printf("Hello from thread:%d in block:%d thread%d \n",index,blockIdx.x,threadIdx.x);
}

void hello_GPU(){
    dim3 grid_1D={2,1,1};
    dim3 block_1D={32,1,1};
    hello_GPU_1D<<<grid_1D,block_1D>>>();
    hipDeviceSynchronize();
}