#include <hip/hip_runtime.h>
#include <stdio.h>



__global__ void hello_GPU_1D(){
    int index=threadIdx.x+blockIdx.x*blockDim.x;
    printf("Hello from thread:%d in block:%d thread%d \n",index,blockIdx.x,threadIdx.x);
}

int main(){
    dim3 grid_1D={2,1,1};
    dim3 block_1D={4,1,1};
    hello_GPU_1D<<<grid_1D,block_1D>>>();
    hipDeviceSynchronize();
    return 0;
}