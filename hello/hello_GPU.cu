#include "hip/hip_runtime.h"
﻿#include "hello_GPU.h"
#define SIZE 2
#define BLOCKSIZE SIZE*SIZE*SIZE*SIZE


__global__ void hello_GPU_2D(int* block){
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    int idy=blockIdx.y*blockDim.y+threadIdx.y;

    int index=gridDim.x*blockDim.x*idy+idx;
    printf("Hello from thread:%d as number %d in block:(%d,%d) thread(%d,%d) \n"
        ,index,block[index],blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
}

void hello_GPU(){
    int block[BLOCKSIZE*BLOCKSIZE]={0};
    for(int i=0;i<BLOCKSIZE*BLOCKSIZE;i++){
        block[i]=i;
    }
    int * d_ptr=nullptr;
    hipMalloc((void**)&d_ptr,BLOCKSIZE*BLOCKSIZE*sizeof(int));
    hipMemcpy(d_ptr,block,BLOCKSIZE*BLOCKSIZE*sizeof(int),hipMemcpyDefault);

    dim3 grid_1D={SIZE,SIZE};
    dim3 block_1D={BLOCKSIZE*SIZE,SIZE};
    hello_GPU_2D<<<grid_1D,block_1D>>>(d_ptr);
    hipDeviceSynchronize();
    hipFree(d_ptr);
}
