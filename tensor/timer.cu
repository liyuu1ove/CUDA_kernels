﻿#include <hip/hip_runtime.h>
#include "../hello/hello_GPU.h"


int main(){
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start); 
hello_GPU(); 
hipEventRecord(stop); 
hipEventSynchronize(stop);
float time_ms;
hipEventElapsedTime(&time_ms, start, stop);
printf("Kernel time: %f ms\n", time_ms);

hipEventDestroy(start);
hipEventDestroy(stop);
}